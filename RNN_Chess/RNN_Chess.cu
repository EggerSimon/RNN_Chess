#include "RNN_Chess.cuh"

//Constructor used to allocate memory on the graphics card
RNN_Chess::RNN_Chess(int Dimensions[])
{
	variables.AllocateWorkspace(Dimensions);
	layerCalculation.InitializeVariables(variables);
	evaluation.KernelSize(Dimensions[1] * Dimensions[3]);
}

//Initializes the learning rate
int RNN_Chess::InitializeConstants(float learningrate)
{
	variables.InitializeConstants(learningrate);
	return 0;
}

//Initializes the Weight Matrices
int RNN_Chess::InitializeVariables(float** InputWeights, float** RecurrentWeights, float** Biases)
{
	variables.InitializeVariables(InputWeights, RecurrentWeights, Biases);
	return 0;
}

//Runs recurrent nerual net
float* RNN_Chess::RunRNN(float* InputState)
{
	char** descriptions = new char*[4]{ "ERR_FORGET_FORWARD" ,"ERR_INPUT_FORWARD" ,"ERR_OUTPUT_FORWARD" ,"ERR_CELL_FORWARD" };
	float*** Gates = new float**[4]{ variables.d_ForgetGate, variables.d_InputGate, variables.d_OutputGate, variables.d_CellGate };

	hipError_t error = hipMemcpy(variables.d_InputStates[variables.h_StateCount], InputState, variables.h_Dimensions[2] * sizeof(float), hipMemcpyHostToDevice);
	variables.CheckCudaError(error, "ERR_VAR_INIT (InputState)");

	//Scales InputMatrix to [-1,1] => mean = 0
	inputScaling.ScaleInput(variables);

	//for each stacked LSTM block
	for (int i = 0; i < variables.h_Dimensions[3]; i++)
	{
		//for each gate
		for (int j = 0; j < 4; j++)
		{
			gateCalculations.GateCalculation(Gates[j][variables.h_StateCount], j, i, descriptions[j], variables);
		}
		layerCalculation.StateCalculation(i, variables);
	}

	variables.GetResults();
	variables.h_StateCount++;

	return variables.h_Results;
}

int RNN_Chess::ErrorCalculation(int color)
{
	hipError_t error;
	char** descriptions = new char*[4]{ "ERR_FORGET_ERRORCALCULATION" ,"ERR_INPUT_ERRORCALCULATION" ,"ERR_OUTPUT_ERRORCALCULATION" ,"ERR_CELL_ERRORCALCULATION" };
	float*** ErrorGates = new float**[4]{ variables.d_Error_ForgetGate, variables.d_Error_InputGate, variables.d_Error_OutputGate, variables.d_Error_CellGate };

	//Sets last Inputstate to detect the end of the match
	if (variables.h_StateCount == variables.h_Dimensions[0])
	{
		inputScaling.setLastInput(variables, color);

		error = hipMemset(variables.d_Error_HiddenStates[variables.h_StateCount], 0, variables.h_Dimensions[3] * 64 * sizeof(float));
		variables.CheckCudaError(error, "ERR_MEMSET");

		color = -1;
	}

	variables.h_StateCount--;

	error = hipMemset(variables.d_Error_HiddenStates[variables.h_StateCount], 0, variables.h_Dimensions[3] * 64 * sizeof(float));
	variables.CheckCudaError(error, "ERR_MEMSET");

	layerCalculation.GetStateError(color, variables, evaluation);

	for (int i = 0; i < variables.h_Dimensions[3]; i++)
	{
		layerCalculation.UpdateGates(variables.h_Dimensions[3] - i - 1, variables);

		for (int j = 0; j < 4; j++)
		{
			gateCalculations.BackwardPass(ErrorGates[j], variables.h_Dimensions[3] - i - 1, j, descriptions[j], variables);
		}
	}

	return 0;
}

int RNN_Chess::BackPropagation()
{
	char** descriptions = new char*[4]{ "ERR_FORGET_BACKWARD" ,"ERR_INPUT_BACKWARD" ,"ERR_OUTPUT_BACKWARD" ,"ERR_CELL_BACKWARD" };
	float*** ErrorGates = new float**[4]{ variables.d_Error_ForgetGate, variables.d_Error_InputGate, variables.d_Error_OutputGate, variables.d_Error_CellGate };

	for (int i = 0; i < variables.h_Dimensions[0]; i++) {
		for (int j = 0; j < variables.h_Dimensions[3]; j++) {
			for (int k = 0; k < 4; k++) {
				gateCalculations.UpdateGates(ErrorGates[k], j, k, i, descriptions[k], variables);
			}
		}
	}

	return 0;
}

//Updates host weight variables
int RNN_Chess::UpdateWeightMatrices(float** InputWeights, float** RecurrentWeights, float** Biases)
{
	variables.UpdateWeightMatrices(InputWeights, RecurrentWeights, Biases);
	return 0;
}

void RNN_Chess::UpdateDimensions(int Dimensions[])
{
	hipError_t error;

	variables.h_Dimensions = new int[6];

	for (int i = 0; i < 6; i++)
	{
		variables.h_Dimensions[i] = Dimensions[i];
	}

	if (variables.h_StateCount != 0)
	{
		std::cout << "ERR_CALCULATION" << std::endl;
		variables.h_StateCount = 0;
	}
	evaluation.UpdateEpoch(&variables);
}

//Frees the before needed workspace
int RNN_Chess::FreeWorkSpace()
{
	evaluation.UpdateEpoch(&variables);

	variables.FreeWorkspace();
	return 0;
}



